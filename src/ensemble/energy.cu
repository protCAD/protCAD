#include "hip/hip_runtime.h"
#include "energy.h"

// pointer to data copied to device
double* x_d; 
double* y_d; 
double* z_d; 
double* rad_d; 
double* eps_d; 
double* chg_d;
double* vol_d; 
double* dis_d;
int* bon_d;
int* clash_d;
double* E_d;

// Constants on device
__constant__ double watRad = 1.4; 
__constant__ double watDia = 4.35; //effective diameter between water molecules
__constant__ double watVol = 107.31; // vol of effective diameter
__constant__ double watPol = 1.47; // water polarization (Murphy WF. J. Chem. Phys. 1977;67:5877–5882)
__constant__ double watEps = 0.15200; 
__constant__ double pi = 3.1415926535;
__constant__ double kc = 332.0636;
__constant__ double kb = 0.0019872041;
__constant__ double t = 300.0;
__constant__ double v = 4.188; // 4/3*pi

// Error checking
inline void _check(hipError_t code, const char *file, int line)
{
  if (code != hipSuccess) {
    fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);}
}

///////////Kernels//////////////////////////////////////////////////////////////////

// Kernel function for calculating distances and atom environment volumes
__global__ void calcDistance(double* x, double* y, double* z, double* rad, double* vol, double* dist, int N)
{
  // Compute the global thread index
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute the indices of the two points for this thread
  int i = idx / N;
  int j = idx % N;
  int index = i*(N-1)-(i-1)*i/2+j-i-1;

  if (j > i){
    // Calculate distance between i and j
    double distance = sqrt((x[i]-x[j])*(x[i]-x[j]) + (y[i]-y[j])*(y[i]-y[j]) + (z[i]-z[j])*(z[i]-z[j]));

    // Estimate volume around atoms occupied by other atoms
    // exposure to solvent can be estimated without a surface calc by estimating cavity volume around atom
    double volJ = v*pow(rad[j],3); double volI = v*pow(rad[i],3);
    if (distance < rad[i]+watDia) {atomicAdd(&vol[i], volJ);}
    if (distance < rad[j]+watDia) {atomicAdd(&vol[j], volI);}

    // save dist to array 
    dist[index] = distance;
  }
}

// Kernel function for calculating distances and atom environment volumes
__global__ void calcClash(double* x, double* y, double* z, double* rad, int* bon, int* clash, int N)
{
  // Compute the global thread index
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute the indices of the two points for this thread
  int i = idx / N;
  int j = idx % N;
  int index = i*(N-1)-(i-1)*i/2+j-i-1;

  if (j > i){
    if (bon[index] == 0){
      // Calculate distanceSquared between i and j
      double distanceSq = (x[i]-x[j])*(x[i]-x[j]) + (y[i]-y[j])*(y[i]-y[j]) + (z[i]-z[j])*(z[i]-z[j]);

      //count clashes between i and j
      if (distanceSq < (rad[i]+rad[j])*(rad[i]+rad[j])) {
        //printf("%d\n",clash[i]);
        atomicAdd(&clash[i],1); atomicAdd(&clash[j],1);
      }
    }
  }
}

// Kernel function for calculating the energy between two atoms
__global__ void calcEnergy(double* rad, double* eps, double* chg, double* vol, double* dist, int* bon, double *E, int N)
{
  // Compute the global thread index
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Compute the indices of the two points for this thread
  int i = idx / N;
  int j = idx % N;
  int index = i*(N-1)-(i-1)*i/2+j-i-1;
  

  if (j > i){
    if (bon[index] == 0){
      //Calculate VDW Energies
      double vdw = (sqrt(eps[i]*eps[j])) * (pow(((rad[i]+rad[j])/dist[index]),12) - (2*pow(((rad[i]+rad[j])/dist[index]),6)));
      atomicAdd(E, vdw);

      // Estimate the number of waters occupying remaining shell volume of i to be used for local dielectric
      double polI=0.0; double watersI = 0.0;
      double shellVolI = v*pow((rad[i]+watDia),3);
      double envVolI = (vol[i]+(v*pow(rad[i],3)))/2;
      double waterVolI = shellVolI-envVolI;
      if (waterVolI > 0){watersI = int(waterVolI/watVol); polI = watersI*watPol;}

      // Estimate the number of waters occupying remaining shell volume of j to be used for local dielectric
      double polJ=0.0; double watersJ = 0.0;
      double shellVolJ = v*pow((rad[j]+watDia),3);
      double envVolJ = (vol[j]+(v*pow(rad[j],3)))/2;
      double waterVolJ = shellVolJ-envVolJ;
      if (waterVolJ > 0){watersJ = int(waterVolJ/watVol); polJ = watersJ*watPol;}

      //Calculate the effective dielectric with the Lorentz local field correction
      double dielectricI =2+(8*pi/3)*(polI)/1-(4*pi/3)*(polI);
      double dielectricJ =2+(8*pi/3)*(polJ)/1-(4*pi/3)*(polJ);
      double dielectric = (dielectricI+dielectricJ)/2;
      
      //Calculate Electrostatic Energies
      double ele = (kc * (chg[i] * chg[j]) / dist[index]) / dielectric;
      atomicAdd(E, ele);
    }
  }
  // Calculate solvation energy of each atom (i)
  if (i == j){

    // Estimate the number of waters occupying remaining shell volume of i to be used for solvation
    double shellVolI = v*pow((rad[i]+watDia),3);
    double envVolI = (vol[i]+(v*pow(rad[i],3)))/2;
    double waterVolI = shellVolI-envVolI;
    if (waterVolI > 0){
      double watersI = int(waterVolI/watVol); double polI = watersI*watPol;
    
      //Calculate the effective dielectric with the Lorentz local field correction
      double dielectricI =2+(8*pi/3)*(polI)/1-(4*pi/3)*(polI);
      
      // Calculate electrostatic solvation using born approximation
      double eleSolv = -(kc/2)*(chg[i]*chg[i])/((rad[i]+watRad)*dielectricI)*watersI;
      atomicAdd(E, eleSolv);

      // Calculate ideal vdw interactions with waters
      double vdwSolv = (sqrt(eps[i]*watEps) * -1)*watersI;
      atomicAdd(E, vdwSolv);

      // Calculate hydrophobic effect (water entropy cost)
      double entSolv = -(kb*t)*log(pow(0.5,watersI));
      atomicAdd(E, entSolv);
    }
  }
}
///////////Kernels End//////////////////////////////////////////////////////////////////


//////////Memory allocation////////////////////////////////////////////////////////////

void loadEnergyDeviceMem(double* x_h, double* y_h, double* z_h, double* rad_h, double* eps_h, double* chg_h, double* vol_h, int* bon_h, double *E_h, int N)
{
  // Allocate memory on the GPU for the arrays
  int bondingSize = N * (N - 1) / 2;
  check(hipMalloc(&x_d,   N * sizeof(double))); check(hipMalloc(&y_d,   N * sizeof(double))); check(hipMalloc(&z_d,   N * sizeof(double)));
  check(hipMalloc(&rad_d, N * sizeof(double))); check(hipMalloc(&eps_d, N * sizeof(double))); check(hipMalloc(&chg_d, N * sizeof(double)));
  check(hipMalloc(&vol_d, N * sizeof(double))); check(hipMalloc(&bon_d, bondingSize * sizeof(int))); 
  check(hipMalloc(&dis_d, bondingSize * sizeof(double))); check(hipMalloc(&E_d, sizeof(double)));
  
  // Copy the coordinates, radius, epsilon, charge and bonding from the host (CPU) to the device (GPU)
  check(hipMemcpy(x_d, x_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(y_d, y_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(z_d, z_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(rad_d, rad_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(eps_d, eps_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(chg_d, chg_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(vol_d, vol_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(bon_d, bon_h, bondingSize * sizeof(int), hipMemcpyHostToDevice));
  check(hipMemcpy(E_d, E_h, sizeof(double), hipMemcpyHostToDevice));
}

void loadClashDeviceMem(double* x_h, double* y_h, double* z_h, double* rad_h, int* bon_h, int* clash_h, int N)
{
  // Allocate memory on the GPU for the arrays
  int bondingSize = N * (N - 1) / 2;
  check(hipMalloc(&x_d,   N * sizeof(double))); check(hipMalloc(&y_d,   N * sizeof(double))); check(hipMalloc(&z_d,   N * sizeof(double)));
  check(hipMalloc(&rad_d, N * sizeof(double))); check(hipMalloc(&clash_d, N * sizeof(int))); check(hipMalloc(&bon_d, bondingSize * sizeof(int))); 
  
  // Copy the coordinates, radius, epsilon, charge and bonding from the host (CPU) to the device (GPU)
  check(hipMemcpy(x_d, x_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(y_d, y_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(z_d, z_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(rad_d, rad_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(clash_d, clash_h, N * sizeof(int), hipMemcpyHostToDevice));
  check(hipMemcpy(bon_d, bon_h, bondingSize * sizeof(int), hipMemcpyHostToDevice));
}

void loadAllDeviceMem(double* x_h, double* y_h, double* z_h, double* rad_h, double* eps_h, double* chg_h, double* vol_h, int* clash_h, int* bon_h, double *E_h, int N)
{
  // Allocate memory on the GPU for the arrays
  int bondingSize = N * (N - 1) / 2;
  check(hipMalloc(&x_d,   N * sizeof(double))); check(hipMalloc(&y_d,   N * sizeof(double))); check(hipMalloc(&z_d,   N * sizeof(double)));
  check(hipMalloc(&rad_d, N * sizeof(double))); check(hipMalloc(&eps_d, N * sizeof(double))); check(hipMalloc(&chg_d, N * sizeof(double)));
  check(hipMalloc(&vol_d, N * sizeof(double))); check(hipMalloc(&clash_d, N * sizeof(int))); check(hipMalloc(&bon_d, bondingSize * sizeof(int))); 
  check(hipMalloc(&dis_d, bondingSize * sizeof(double))); check(hipMalloc(&E_d, sizeof(double)));
  
  // Copy the coordinates, radius, epsilon, charge and bonding from the host (CPU) to the device (GPU)
  check(hipMemcpy(x_d, x_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(y_d, y_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(z_d, z_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(rad_d, rad_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(eps_d, eps_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(chg_d, chg_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(vol_d, vol_h, N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(clash_d, clash_h, N * sizeof(int), hipMemcpyHostToDevice));
  check(hipMemcpy(bon_d, bon_h, bondingSize * sizeof(int), hipMemcpyHostToDevice));
  check(hipMemcpy(E_d, E_h, sizeof(double), hipMemcpyHostToDevice));
}

void freeEnergyDeviceMem()
{
  // Free the GPU memory
  check(hipFree(x_d)); check(hipFree(y_d)); check(hipFree(z_d));
  check(hipFree(rad_d)); check(hipFree(eps_d)); check(hipFree(chg_d));
  check(hipFree(bon_d)); check(hipFree(vol_d)); check(hipFree(dis_d)); check(hipFree(E_d));
}

void freeClashDeviceMem()
{
  // Free the GPU memory
  check(hipFree(x_d)); check(hipFree(y_d)); check(hipFree(z_d));
  check(hipFree(rad_d)); check(hipFree(clash_d)); check(hipFree(bon_d)); 
}

void freeAllDeviceMem()
{
  // Free the GPU memory
  check(hipFree(x_d)); check(hipFree(y_d)); check(hipFree(z_d));
  check(hipFree(rad_d)); check(hipFree(eps_d)); check(hipFree(chg_d)); check(hipFree(clash_d));
  check(hipFree(bon_d)); check(hipFree(vol_d)); check(hipFree(dis_d)); check(hipFree(E_d));
}
///////////Memory allocation End///////////////////////////////////////////////////////


//////////Functions///////////////////////////////////////////////////////////////////

void calcEnergies(double* x_h, double* y_h, double* z_h, double *E_h, int N)
{
  // Update the coordinates and starting energy from the host to the GPU
  check(hipMemcpy(x_d, x_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(y_d, y_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(z_d, z_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(E_d, E_h, sizeof(double), hipMemcpyHostToDevice));
  
  // Bound and invoke the distance kernel
  int tC = N * (N-1) / 2;
  int blocks = (tC+threads_per_block-1)/threads_per_block;
  calcDistance <<< blocks, threads_per_block >>> (x_d,y_d,z_d,rad_d,vol_d,dis_d,N);
  calcEnergy <<< blocks, threads_per_block >>> (rad_d,eps_d,chg_d,vol_d,dis_d,bon_d,E_d,N);
  //check(hipPeekAtLastError());
  check(hipDeviceSynchronize());
  
  // Copy the final energy back to the host
  check(hipMemcpy(E_h, E_d, sizeof(double), hipMemcpyDeviceToHost));
}

void calcClashes(double* x_h, double* y_h, double* z_h, int* clash_h, int N)
{
  // Update the coordinates and starting energy from the host to the GPU
  check(hipMemcpy(x_d, x_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(y_d, y_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(z_d, z_h,     N * sizeof(double), hipMemcpyHostToDevice));
  check(hipMemcpy(clash_d, clash_h, N * sizeof(int), hipMemcpyHostToDevice));
  
  // Bound and invoke the distance kernel
  int tC = N * (N-1) / 2;
  int blocks = (tC+threads_per_block-1)/threads_per_block;
  calcClash <<< blocks, threads_per_block >>> (x_d,y_d,z_d,rad_d,bon_d,clash_d,N);
  //check(hipPeekAtLastError());
  check(hipDeviceSynchronize());
  
  // Copy the final energy back to the host
  check(hipMemcpy(clash_h, clash_d, N * sizeof(int), hipMemcpyDeviceToHost));
}

//////////Functions end///////////////////////////////////////////////////////////////

